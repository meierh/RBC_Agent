#include "hip/hip_runtime.h"
#include "chessinformationset.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

namespace crazyara {

void CHECK(hipError_t cuError)
{
    if(cuError!=hipSuccess)
    {   
        std::string cudaErrorString(hipGetErrorString(cuError));
        std::cout<<"CUDA Error: "<<cudaErrorString<<std::endl;
        throw std::logic_error("CUDA Fail");
    }
}

__device__ bool evaluateLiteral
(
    uint8_t* literal,
    uint8_t* board
)
{
    uint8_t sum = 0;
    for(uint8_t byteInd=0; byteInd<48; byteInd++)
    {
        sum += *(literal+byteInd) & *(board+byteInd);
        //printf("lit:%d & board:%d = %d\n",*(literal+byteInd),*(board+byteInd),*(literal+byteInd) & *(board+byteInd));
    }
    return (sum!=0)?true:false;
}

__device__ bool CISgetBit
(
    uint8_t byte,
    uint8_t position
)
{
    return byte & (1 << position);
}

__global__ void checkConditions
(
    uint8_t numberOfConditions,
    uint8_t* clausesPerCondition,
    uint8_t* conditionArray,
    uint8_t* boardInfoSet,
    uint64_t boardInfoSetSize,
    uint8_t* incompatibleBoards    
)
{
    __shared__ uint8_t clauseNbr[25];
    __shared__ uint8_t clauses[25][2][49];
    uint8_t* clauseStart = conditionArray;
    if(threadIdx.x==0)
    {
        for(uint condInd=0; condInd<numberOfConditions; condInd++)
        {
            uint8_t numberOfClauses = clausesPerCondition[condInd];
            clauseNbr[condInd] = numberOfClauses;
            for(uint clauseInd=0; clauseInd<numberOfClauses; clauseInd++)
            {
                for(uint byteInd=0; byteInd<49; byteInd++)
                {
                    clauses[condInd][clauseInd][byteInd] = *(clauseStart+byteInd);
                }
                clauseStart += 49;
            }
        }
    }
    __syncthreads();
    
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < boardInfoSetSize)
    {
        uint8_t board[58];
        for(uint8_t byteInd=0; byteInd<58; byteInd++)
        {
            board[byteInd] = boardInfoSet[index*58+byteInd];
        }
        
        bool compatible = true;
        for(uint8_t condInd=0; condInd<numberOfConditions; condInd++)
        {
            bool oneClauseTrue = false;
            uint8_t thisCondClauseNbr = clauseNbr[condInd];
            //printf("oneClauseTrue %d\n",oneClauseTrue);
            // IMPORTANT: No clause can be be empty
            for(uint8_t clauseInd=0; clauseInd<thisCondClauseNbr; clauseInd++)
            {
                bool clauseDemandBool = (bool)clauses[condInd][clauseInd][0];
                uint8_t* clauseBits = clauses[condInd][clauseInd]+1;
                bool literalBool = evaluateLiteral(clauseBits,board);
                oneClauseTrue |= (clauseDemandBool==literalBool);
                //printf("oneClauseTrue %d\n",oneClauseTrue);
            }
            compatible &= oneClauseTrue;
        }
        //printf("compatible %d\n",compatible);
        incompatibleBoards[index] = (compatible)?1:0;
    }
}

void ChessInformationSet::markIncompatibleBoardsGPU
(
    const std::vector<BoardClause>& conditions
)
{
    std::cout<<"Mark boards that do not fit: ";
    for(auto clause : conditions)
        std::cout<<clause.to_string()<<"&&";
    std::cout<<std::endl;
    
    std::unique_ptr<std::vector<std::uint8_t>> incompatibleBoard = checkBoardsValidGPU(conditions);
    // This is wrong
    std::for_each(incompatibleBoard->begin(),incompatibleBoard->end(),
                  [&](std::uint8_t boardIndex){incompatibleBoards.push(boardIndex);});
}

std::unique_ptr<std::vector<std::uint8_t>> ChessInformationSet::checkBoardsValidGPU
(
    const std::vector<BoardClause>& conditions
)
{
    /*
    std::cout<<"Mark boards that do not fit: ";
    for(auto clause : conditions)
        std::cout<<clause.to_string()<<"&&";
    std::cout<<std::endl;
    */
    
    std::vector<std::vector<std::pair<std::uint8_t,std::array<std::uint8_t,48>>>> hostBitwiseCondition;
    hostBitwiseCondition.resize(conditions.size());
    std::uint8_t numberOfConditions = conditions.size();
    std::vector<std::uint8_t> hostClausesPerCondition(numberOfConditions);
    if(numberOfConditions>25)
        throw std::logic_error("There must maximal 25 conditions");
    for(uint conditionInd=0; conditionInd<numberOfConditions; conditionInd++)
    {
        conditions[conditionInd].to_bits(hostBitwiseCondition[conditionInd]);
        hostClausesPerCondition[conditionInd] = hostBitwiseCondition[conditionInd].size();
        if(hostBitwiseCondition[conditionInd].size()>2)
            throw std::logic_error("There must only be 2 clauses per condition");
    }
    
    //std::cout<<"numberOfConditions:"<<int(numberOfConditions)<<std::endl;
    
    uint8_t* deviceClausesPerCondition;
    CHECK(hipMalloc((void**)&deviceClausesPerCondition,numberOfConditions*sizeof(uint8_t)));
    CHECK(hipMemcpy(deviceClausesPerCondition,hostClausesPerCondition.data(),
                         numberOfConditions*sizeof(uint8_t),hipMemcpyHostToDevice));
    
    //std::cout<<"Start remapping"<<std::endl;
    std::vector<std::uint8_t> remapClausesPerCondition(numberOfConditions);
    CHECK(hipMemcpy(remapClausesPerCondition.data(),deviceClausesPerCondition,
                         numberOfConditions*sizeof(uint8_t),hipMemcpyDeviceToHost));
    //std::for_each(remapClausesPerCondition.begin(),remapClausesPerCondition.end(),[](auto ind){std::cout<<int(ind)<<" ";});
    //std::cout<<"Remapped"<<std::endl;
    
    std::vector<std::pair<std::uint8_t,std::array<std::uint8_t,48>>> hostBitwiseConditionFlat;
    for(auto oneCondition : hostBitwiseCondition)
    {
        /*
        for(auto oneClause : oneCondition)
        {
            auto firstPart = oneClause.first;
            for(auto byte : firstPart)
                std::cout<<uint(byte)<<" ";
            std::cout<<std::endl;
            auto secondPart = oneClause.second;
            for(auto byte : secondPart)
                std::cout<<uint(byte)<<" ";
            std::cout<<std::endl;
        }
        */
        hostBitwiseConditionFlat.insert(hostBitwiseConditionFlat.end(),oneCondition.begin(),oneCondition.end());
    }
    
    uint8_t* deviceBitwiseCondition;
    uint byteSizeMem = hostBitwiseConditionFlat.size()*sizeof(uint8_t)*49;
    CHECK(hipMalloc((void**)&(deviceBitwiseCondition),byteSizeMem));
    CHECK(hipMemcpy(deviceBitwiseCondition,hostBitwiseConditionFlat.data(),byteSizeMem,hipMemcpyHostToDevice));    

    //std::cout<<"Start remapping"<<std::endl;
    std::vector<std::pair<std::uint8_t,std::array<std::uint8_t,48>>> remapHostBitwiseCondition(hostBitwiseConditionFlat.size());
    CHECK(hipMemcpy(remapHostBitwiseCondition.data(),deviceBitwiseCondition,byteSizeMem,hipMemcpyDeviceToHost));    
    
    /*
    for(auto oneCondition : remapHostBitwiseCondition)
    {
        std::cout<<"Cond:"<<std::endl;
        auto firstPart = oneCondition.first;
        std::cout<<" Must be: ";
        for(auto byte : firstPart)
            std::cout<<uint(byte)<<" ";
        std::cout<<std::endl;
        auto secondPart = oneCondition.second;
        std::cout<<" Must not be: ";
        for(auto byte : secondPart)
            std::cout<<uint(byte)<<" ";
        std::cout<<std::endl;
    }
    std::cout<<"Remapped"<<std::endl;
    */
    

    
    std::uint64_t cis_size = size();
    std::uint64_t cis_byte_size = cis_size*(chessInfoSize/8);
    std::uint8_t* hostInfoSetPtr = getInfoSetPtr();
    uint8_t* deviceInfoSetPtr;
    CHECK(hipMalloc((void**)&deviceInfoSetPtr,cis_byte_size*sizeof(uint8_t)));
    CHECK(hipMemcpy(deviceInfoSetPtr,hostInfoSetPtr,cis_byte_size*sizeof(uint8_t),hipMemcpyHostToDevice));
    
    //std::cout<<"cis_size:"<<int(cis_size)<<std::endl;
    //std::cout<<"cis_byte_size:"<<int(cis_byte_size)<<std::endl;
    
    auto result = std::make_unique<std::vector<std::uint8_t>>(cis_size);
    std::vector<std::uint8_t>& hostIncompatibleBoards = *result;
    uint8_t* deviceIncompatibleBoards;
    CHECK(hipMalloc((void**)&deviceIncompatibleBoards,cis_size*sizeof(uint8_t)));
    
    //std::cout<<"Before kernel invocation"<<std::endl;
    
    int suggested_blockSize; 
    int suggested_minGridSize;
    hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, checkConditions, 0, 0);
    int device;
    hipGetDevice(&device); 
    struct hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);    
    
    //std::cout<<"suggested_blockSize:"<<int(suggested_blockSize)<<std::endl;
    //std::cout<<"suggested_minGridSize:"<<int(suggested_minGridSize)<<std::endl;
    //std::cout<<"device:"<<int(device)<<std::endl;

    dim3 blocks(suggested_blockSize);
    //std::cout<<"blocks.x:"<<blocks.x<<std::endl;
    dim3 grids(ceil((float)cis_size/suggested_blockSize));
    //std::cout<<"grids.x:"<<grids.x<<std::endl;
    
    checkConditions<<<grids,blocks>>>
    (
        numberOfConditions,
        deviceClausesPerCondition,
        deviceBitwiseCondition,
        deviceInfoSetPtr,
        cis_size,
        deviceIncompatibleBoards
    );
    
    //std::cout<<"After kernel invocation"<<std::endl;
    
    CHECK(hipMemcpy(hostIncompatibleBoards.data(),deviceIncompatibleBoards,
                     cis_size*sizeof(uint8_t),hipMemcpyDeviceToHost));
    
    hipFree(deviceClausesPerCondition);
    hipFree(deviceBitwiseCondition);
    hipFree(deviceInfoSetPtr);
    hipFree(deviceIncompatibleBoards);

    for(uint64_t boardIndex=0; boardIndex<hostIncompatibleBoards.size(); boardIndex++)
    {
        if(hostIncompatibleBoards[boardIndex]==1)
            incompatibleBoards.push(boardIndex);
    }
    
    //std::cout<<"End of function"<<std::endl;

    return result;
}

__global__ void initialReduceDistr // blockDim.x == 32
(
    uint8_t* boardInfoSet,
    uint64_t boardInfoSetSize,
    uint32_t* boardSum //gridsize * (64*6)
)
{
    __shared__ uint32_t distro[32][6][64];
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        for(uint8_t squareInd=0; squareInd<64; squareInd++)
        {
            distro[threadIdx.x][pieceInd][squareInd] = 0;
        }
    }
    __syncthreads();
    uint64_t boardSize = 58;
    uint64_t blockSpan = ceilf((float)boardInfoSetSize / gridDim.x);
    uint64_t blockOffset = blockIdx.x*blockSpan;
    
    // Reduce to 32 boards shared memory
    uint8_t* blockStartPtr = boardInfoSet+blockOffset*boardSize;
    uint64_t validBlockSpan = min(blockSpan,boardInfoSetSize-blockOffset);
    for(uint64_t locIndex=threadIdx.x; locIndex<validBlockSpan; locIndex+=blockDim.x)
    {
        uint8_t* boardStart = blockStartPtr + locIndex*boardSize;
        uint8_t* probabilityPtr = boardStart+57;
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            uint8_t* pieceBoardStart = boardStart + pieceInd*8;
            uint8_t adding = 1; //*probabilityPtr & 127; // unset first bit
            for(uint8_t row=0; row<8; row++)
            {
                uint8_t* boardRow = pieceBoardStart + row;
                for(uint8_t col=0; col<8; col++)
                {
                    bool squareOccupied = CISgetBit(*boardRow,7-col);
                    distro[threadIdx.x][pieceInd][row*8+col] += (squareOccupied)?adding:0; 
                }
            }
        }
    }
    __syncthreads();
    
    for(uint8_t size=1; size<32; size++)
    {
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            distro[0][pieceInd][threadIdx.x] += distro[size][pieceInd][threadIdx.x];
            distro[0][pieceInd][threadIdx.x+32] += distro[size][pieceInd][threadIdx.x+32];
        }
    }
    __syncthreads();
    
    uint32_t* boardSumOffset = boardSum + blockIdx.x*(64*6);
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        *(boardSumOffset + (pieceInd*64) + threadIdx.x) = distro[0][pieceInd][threadIdx.x];
        *(boardSumOffset + (pieceInd*64) + threadIdx.x+32) = distro[0][pieceInd][threadIdx.x+32];
    }
}

__global__ void reduceDistr // blockDim.x == 32
(
    
    uint32_t* boardsIn,
    uint32_t boardInSize,
    uint32_t* boardsOut //gridsize * (64*6)
)
{
    __shared__ uint32_t distro[32][6][64];
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        for(uint8_t squareInd=0; squareInd<64; squareInd++)
        {
            distro[threadIdx.x][pieceInd][squareInd] = 0;
        }
    }
    uint64_t blockSpan = ceilf((float)boardInSize / gridDim.x);
    uint64_t blockOffset = blockIdx.x*blockSpan;
    uint64_t blockOffsetData = blockOffset*6*64;

    for(uint64_t locIndex = 0; locIndex+threadIdx.x<blockSpan; locIndex+=blockDim.x)
    {
        uint32_t* boardStart = boardsIn + blockOffsetData + (locIndex+threadIdx.x)*6*64;
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            for(uint8_t squareInd=0; squareInd<64; squareInd++)
            {
                distro[threadIdx.x][pieceInd][squareInd] += *(boardStart+pieceInd*64+squareInd);
            }
        }
    }
    
    __syncthreads();
    for(uint8_t size=1; size<32; size++)
    {
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            distro[0][pieceInd][threadIdx.x] += distro[size][pieceInd][threadIdx.x];
            distro[0][pieceInd][threadIdx.x+32] += distro[size][pieceInd][threadIdx.x+32];
        }
    }
    
    __syncthreads();
    uint32_t* boardSumOffset = boardsOut + blockIdx.x*(64*6);
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        *(boardSumOffset + (pieceInd*64) + threadIdx.x) = distro[0][pieceInd][threadIdx.x];
        *(boardSumOffset + (pieceInd*64) + threadIdx.x+32) = distro[0][pieceInd][threadIdx.x+32];
    }
}

std::unique_ptr<ChessInformationSet::Distribution> ChessInformationSet::computeDistributionGPU()
{
    //std::cout<<"Compute Distribution"<<std::endl;
    std::uint64_t cis_size = size();
    std::uint64_t maxSize = 1;
    maxSize = maxSize<<32;
    if(cis_size >= maxSize)
    {
        std::cout<<"cis_size:"<<cis_size<<" > "<<maxSize<<std::endl;
        throw std::invalid_argument("CIS size too big");
    }
    std::uint64_t cis_byte_size = cis_size*(chessInfoSize/8);
    std::uint8_t* hostInfoSetPtr = getInfoSetPtr();
    uint8_t* deviceInfoSetPtr;
    CHECK(hipMalloc((void**)&deviceInfoSetPtr,cis_byte_size*sizeof(uint8_t)));
    CHECK(hipMemcpy(deviceInfoSetPtr,hostInfoSetPtr,cis_byte_size*sizeof(uint8_t),hipMemcpyHostToDevice));
    
    /*
    for(int i=0; i<cis_size; i++)
    {
        for(int k=0; k<58; k++)
            std::cout<<int(hostInfoSetPtr[i*58+k])<<" ";
        std::cout<<std::endl;
    }
    std::cout<<std::endl;
    */
        
    //std::cout<<"cis_size:"<<int(cis_size)<<std::endl;
    //std::cout<<"cis_byte_size:"<<int(cis_byte_size)<<std::endl;
    
    dim3 blocks(32);
    
    std::uint64_t maxNbrBlocks = 60000;
    std::uint64_t boardsPerBlock = cis_size / maxNbrBlocks;
    std::uint64_t boardsPerThread = boardsPerBlock / blocks.x;
    std::uint64_t minBoardsPerThread = 8;
    if(boardsPerThread < minBoardsPerThread)
        boardsPerThread = minBoardsPerThread;    
    dim3 grids(ceil((float)cis_size/(blocks.x*boardsPerThread)));

    //std::cout<<"blocks.x:"<<blocks.x<<std::endl;
    //std::cout<<"grids.x:"<<grids.x<<std::endl;
    
    uint32_t* deviceBoardSumIn;
    CHECK(hipMalloc((void**)&deviceBoardSumIn,grids.x*6*64*sizeof(uint32_t)));
    
    uint32_t* deviceBoardSumOut;
    CHECK(hipMalloc((void**)&deviceBoardSumOut,grids.x*6*64*sizeof(uint32_t)));
    
    initialReduceDistr<<<grids,blocks>>>
    (
        deviceInfoSetPtr,
        cis_size,
        deviceBoardSumIn
    );
    hipDeviceSynchronize();
    //std::cout<<"Initial reduction"<<std::endl;
    
    std::uint64_t inGridSize = grids.x;
    std::uint64_t outGridSize;
    while(inGridSize>1)
    {
        outGridSize = ceil((float)inGridSize / 128);
        grids = dim3(outGridSize);
        //std::cout<<"inGridSize:"<<inGridSize<<std::endl;
        //std::cout<<"outGridSize:"<<outGridSize<<std::endl;
        reduceDistr<<<grids,blocks>>>
        (
            deviceBoardSumIn,
            inGridSize,
            deviceBoardSumOut
        );
        hipDeviceSynchronize();
        inGridSize = outGridSize;
        uint32_t* temp = deviceBoardSumIn;
        deviceBoardSumIn = deviceBoardSumOut;
        deviceBoardSumOut = temp;
    }
    //std::cout<<"Copy result back"<<std::endl;
    std::array<std::uint32_t,384> piecesSum;
    CHECK(hipMemcpy(piecesSum.data(),deviceBoardSumIn,384*sizeof(uint32_t),hipMemcpyDeviceToHost));
    
    /*
    for(auto count : piecesSum)
        std::cout<<count<<" ";
    std::cout<<std::endl;
    */
    
    //std::cout<<"Free"<<std::endl;
    hipFree(deviceInfoSetPtr);
    hipFree(deviceBoardSumIn);
    hipFree(deviceBoardSumOut);
    
    //std::cout<<"Compute Fraction"<<std::endl;
    std::array<double,384> piecesSumDouble;
    for(uint i=0; i<piecesSum.size(); i++)
        piecesSumDouble[i] = static_cast<double>(piecesSum[i]) / cis_size;
    
    //std::cout<<"Compute Distribution"<<std::endl;
    auto piecesDistro = std::make_unique<Distribution>();
    std::memcpy(piecesDistro->pawns.data(),  piecesSumDouble.data(),    64*sizeof(double));
    std::memcpy(piecesDistro->knights.data(),piecesSumDouble.data()+64, 64*sizeof(double));
    std::memcpy(piecesDistro->bishops.data(),piecesSumDouble.data()+128,64*sizeof(double));
    std::memcpy(piecesDistro->rooks.data(),  piecesSumDouble.data()+192,64*sizeof(double));
    std::memcpy(piecesDistro->queens.data(), piecesSumDouble.data()+256,64*sizeof(double));
    std::memcpy(piecesDistro->kings.data(),  piecesSumDouble.data()+320,64*sizeof(double));
    //std::cout<<"Return"<<std::endl;
    return piecesDistro;
}

__global__ void initialReduceEntropy // blockDim.x == 32
(
    float* distribution, // 6*64
    uint8_t* boardInfoSet,
    uint64_t boardInfoSetSize,
    float* squareEntropy, //gridsize * (64)
    float* scanSquareEntropy //gridsize * (36)
)
{
    __shared__ float distributionBoard[7][64];
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        distributionBoard[pieceInd][threadIdx.x] = *(distribution+pieceInd*64+threadIdx.x);
        distributionBoard[pieceInd][threadIdx.x+32] = *(distribution+pieceInd*64+threadIdx.x+32);
    }
    float emptyProb0 = 1;
    float emptyProb32 = 1;
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        emptyProb0 -= distributionBoard[pieceInd][threadIdx.x];
        emptyProb32 -= distributionBoard[pieceInd][threadIdx.x+32];
    }
    distributionBoard[6][threadIdx.x] = emptyProb0;
    distributionBoard[6][threadIdx.x+32] = emptyProb32;
    __syncthreads();

    __shared__ float locSquareEntropy[32][64];
    for(uint8_t squareInd=0; squareInd<64; squareInd++)
    {
        locSquareEntropy[threadIdx.x][squareInd] = 0;
    }
    __shared__ float locScanSquareEntropy[32][36];
    for(uint8_t squareInd=0; squareInd<36; squareInd++)
    {
        locScanSquareEntropy[threadIdx.x][squareInd] = 0;
    }
    __syncthreads();
    

    uint64_t boardSize = 58;
    uint64_t blockSpan = ceilf((float)boardInfoSetSize / gridDim.x);
    uint64_t blockOffset = blockIdx.x*blockSpan;
    
    // Reduce to 32 boards shared memory
    uint8_t* blockStartPtr = boardInfoSet+blockOffset*boardSize;
    uint64_t validBlockSpan = min(blockSpan,boardInfoSetSize-blockOffset);
    for(uint64_t locIndex=threadIdx.x; locIndex<validBlockSpan; locIndex+=blockDim.x)
    {
        uint8_t* boardStart = blockStartPtr + locIndex*boardSize;
        uint8_t* probabilityPtr = boardStart+57;
        uint8_t board[6][8];
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            for(uint8_t row=0; row<8; row++)
            {
                board[pieceInd][row] = *(boardStart + pieceInd*8 + row);
            }
        }

        for(uint8_t row=0; row<8; row++)
        {
            for(uint8_t col=0; col<8; col++)
            {
                //Compute Entropy for one square
                uint8_t linearIndFullBoard = row*8+col;
                uint8_t pieceSubInd = 6;
                for(uint8_t pieceInd=0; pieceInd<6; pieceInd)
                {
                    bool squareOccupied = CISgetBit(board[pieceInd][row],7-col);
                    pieceSubInd = (squareOccupied)?pieceInd:pieceSubInd;
                }
                float prob = distributionBoard[pieceSubInd][linearIndFullBoard];
                float entropy = -prob * log2f(prob);
                locSquareEntropy[threadIdx.x][linearIndFullBoard] += entropy;
                
                //Compute entropy for a scare area
                if(row>0 && row<7 && col>0 && col<7)
                {
                    uint8_t linearIndSenseBoard = (row-1)*6+(col-1);
                    float senseProb = 1;
                    for(uint8_t senseRow = row-1; senseRow<row+2; senseRow++)
                    {
                        for(uint8_t senseCol = col-1; senseCol<col+2; senseCol++)
                        {
                            uint8_t linearIndFullBoard = senseRow*8+senseCol;
                            uint8_t pieceSubInd = 6;
                            for(uint8_t pieceInd=0; pieceInd<6; pieceInd)
                            {
                                bool squareOccupied = CISgetBit(board[pieceInd][senseRow],7-senseCol);
                                pieceSubInd = (squareOccupied)?pieceInd:pieceSubInd;
                            }
                            float prob = distributionBoard[pieceSubInd][linearIndFullBoard];
                            senseProb *= prob;
                        }
                    }
                    float entropy = -senseProb * log2f(senseProb);
                    locScanSquareEntropy[threadIdx.x][linearIndSenseBoard] += entropy;
                }
            }            
        }
    }
    __syncthreads();
    
    for(uint8_t size=1; size<32; size++)
    {
        locSquareEntropy[0][threadIdx.x] +=  locSquareEntropy[size][threadIdx.x];
        locSquareEntropy[0][threadIdx.x+32] +=  locSquareEntropy[size][threadIdx.x+32];
        locScanSquareEntropy[0][threadIdx.x] +=  locScanSquareEntropy[size][threadIdx.x];
        if(threadIdx.x+32<36)
        {
            locScanSquareEntropy[0][threadIdx.x+32] +=  locScanSquareEntropy[size][threadIdx.x+32];
        }
    }
    __syncthreads();
    
    float* squareEntropyOffset = squareEntropy + blockIdx.x*(64);
    *(squareEntropyOffset+threadIdx.x) = locSquareEntropy[0][threadIdx.x];
    *(squareEntropyOffset+threadIdx.x+32) = locSquareEntropy[0][threadIdx.x+32];
    
    float* scanEntropyOffset = scanSquareEntropy + blockIdx.x*(36);
    *(scanEntropyOffset+threadIdx.x) = locScanSquareEntropy[0][threadIdx.x];
    if(threadIdx.x+32<36)
    {
        *(scanEntropyOffset+threadIdx.x+32) = locScanSquareEntropy[0][threadIdx.x+32];
    }
}

__global__ void reduceEntropy // blockDim.x == 32
(
    float* squareEntropyIn,
    float* scanSquareEntropyIn,
    uint32_t inSize,
    float* squareEntropyOut, //gridsize * (64)
    float* scanSquareEntropyOut //gridsize * (36)
)
{
    __shared__ float locSquareEntropy[32][64];
    for(uint8_t squareInd=0; squareInd<64; squareInd++)
    {
        locSquareEntropy[threadIdx.x][squareInd] = 0;
    }
    __shared__ float locScanSquareEntropy[32][36];
    for(uint8_t squareInd=0; squareInd<36; squareInd++)
    {
        locScanSquareEntropy[threadIdx.x][squareInd] = 0;
    }
    __syncthreads();
    
    uint64_t blockSpan = ceilf((float)inSize / gridDim.x);
    uint64_t blockOffset = blockIdx.x*blockSpan;
    uint64_t blockOffsetDataSquareEntropy = blockOffset*64;
    uint64_t blockOffsetDataScanEntropy = blockOffset*36;
    
    for(uint64_t locIndex = 0; locIndex+threadIdx.x<blockSpan; locIndex+=blockDim.x)
    {
        float* squareEntropy = squareEntropyIn + blockOffsetDataSquareEntropy + (locIndex+threadIdx.x)*64;
        for(uint8_t squareInd=0; squareInd<64; squareInd++)
        {
            locSquareEntropy[threadIdx.x][squareInd] += *(squareEntropy + threadIdx.x*64 + squareInd);
        }
        float* scanEntropy = scanSquareEntropyIn + blockOffsetDataScanEntropy + (locIndex+threadIdx.x)*36;
        for(uint8_t squareInd=0; squareInd<36; squareInd++)
        {
            locScanSquareEntropy[threadIdx.x][squareInd] += *(scanEntropy + threadIdx.x*36 + squareInd);
        }
    }
    __syncthreads();
    
    for(uint8_t size=1; size<32; size++)
    {
        locSquareEntropy[0][threadIdx.x] +=  locSquareEntropy[size][threadIdx.x];
        locSquareEntropy[0][threadIdx.x+32] +=  locSquareEntropy[size][threadIdx.x+32];
        locScanSquareEntropy[0][threadIdx.x] +=  locScanSquareEntropy[size][threadIdx.x];
        if(threadIdx.x+32<36)
        {
            locScanSquareEntropy[0][threadIdx.x+32] +=  locScanSquareEntropy[size][threadIdx.x+32];
        }
    }
    __syncthreads();
    
    float* squareEntropyOffset = squareEntropyOut + blockIdx.x*(64);
    *(squareEntropyOffset+threadIdx.x) = locSquareEntropy[0][threadIdx.x];
    *(squareEntropyOffset+threadIdx.x+32) = locSquareEntropy[0][threadIdx.x+32];
    
    float* scanEntropyOffset = scanSquareEntropyOut + blockIdx.x*(36);
    *(scanEntropyOffset+threadIdx.x) = locScanSquareEntropy[0][threadIdx.x];
    if(threadIdx.x+32<36)
    {
        *(scanEntropyOffset+threadIdx.x+32) = locScanSquareEntropy[0][threadIdx.x+32];
    }
}

std::unique_ptr<std::pair<std::array<double,64>,std::array<double,36>>> ChessInformationSet::computeEntropyGPU
(
    const Distribution& hypotheseDistro
)
{
    
    std::array<double,384> distributionfp64;
    std::memcpy(distributionfp64.data(),    hypotheseDistro.pawns.data(),  64*sizeof(double));
    std::memcpy(distributionfp64.data()+64, hypotheseDistro.knights.data(),64*sizeof(double));
    std::memcpy(distributionfp64.data()+128,hypotheseDistro.bishops.data(),64*sizeof(double));
    std::memcpy(distributionfp64.data()+192,hypotheseDistro.rooks.data(),  64*sizeof(double));
    std::memcpy(distributionfp64.data()+256,hypotheseDistro.queens.data(), 64*sizeof(double));
    std::memcpy(distributionfp64.data()+320,hypotheseDistro.kings.data(),  64*sizeof(double));
    
    std::array<float,384> distributionfp32;
    for(uint i=0; i<distributionfp32.size(); i++)
        distributionfp32[i] = distributionfp64[i];
    float* deviceDistribution;
    CHECK(hipMalloc((void**)&deviceDistribution,distributionfp32.size()*sizeof(float)));
    CHECK(hipMemcpy(deviceDistribution,distributionfp32.data(),distributionfp32.size()*sizeof(float),hipMemcpyHostToDevice));    
    
    //std::cout<<"Compute Distribution"<<std::endl;
    std::uint64_t cis_size = size();
    std::uint64_t maxSize = 1;
    maxSize = maxSize<<32;
    if(cis_size >= maxSize)
    {
        std::cout<<"cis_size:"<<cis_size<<" > "<<maxSize<<std::endl;
        throw std::invalid_argument("CIS size too big");
    }
    std::uint64_t cis_byte_size = cis_size*(chessInfoSize/8);
    std::uint8_t* hostInfoSetPtr = getInfoSetPtr();
    uint8_t* deviceInfoSetPtr;
    CHECK(hipMalloc((void**)&deviceInfoSetPtr,cis_byte_size*sizeof(uint8_t)));
    CHECK(hipMemcpy(deviceInfoSetPtr,hostInfoSetPtr,cis_byte_size*sizeof(uint8_t),hipMemcpyHostToDevice));
    
    
    dim3 blocks(32);
    
    std::uint64_t maxNbrBlocks = 60000;
    std::uint64_t boardsPerBlock = cis_size / maxNbrBlocks;
    std::uint64_t boardsPerThread = boardsPerBlock / blocks.x;
    std::uint64_t minBoardsPerThread = 8;
    if(boardsPerThread < minBoardsPerThread)
        boardsPerThread = minBoardsPerThread;    
    dim3 grids(ceil((float)cis_size/(blocks.x*boardsPerThread)));

    //std::cout<<"blocks.x:"<<blocks.x<<std::endl;
    //std::cout<<"grids.x:"<<grids.x<<std::endl;
    
    float* squareEntropyIn;
    CHECK(hipMalloc((void**)&squareEntropyIn,grids.x*64*sizeof(float)));
    
    float* squareEntropyOut;
    CHECK(hipMalloc((void**)&squareEntropyOut,grids.x*64*sizeof(float)));
    
    float* scanSquareEntropyIn;
    CHECK(hipMalloc((void**)&scanSquareEntropyIn,grids.x*36*sizeof(float)));
    
    float* scanSquareEntropyOut;
    CHECK(hipMalloc((void**)&scanSquareEntropyOut,grids.x*36*sizeof(float)));
    
    initialReduceEntropy<<<grids,blocks>>>
    (
        deviceDistribution,
        deviceInfoSetPtr,
        cis_size,
        squareEntropyIn,
        scanSquareEntropyIn
    );
    hipDeviceSynchronize();
    //std::cout<<"Initial reduction"<<std::endl;
    
    std::uint64_t inGridSize = grids.x;
    std::uint64_t outGridSize;
    while(inGridSize>1)
    {
        outGridSize = ceil((float)inGridSize / 128);
        grids = dim3(outGridSize);
        reduceEntropy<<<grids,blocks>>>
        (
            squareEntropyIn,
            scanSquareEntropyIn,
            inGridSize,
            squareEntropyOut,
            scanSquareEntropyOut
        );
        hipDeviceSynchronize();
        inGridSize = outGridSize;
        float* temp;
        
        temp = squareEntropyIn;
        squareEntropyIn = squareEntropyOut;
        squareEntropyOut = temp;
        
        temp = scanSquareEntropyIn;
        scanSquareEntropyIn = scanSquareEntropyOut;
        scanSquareEntropyOut = temp;
    }
    //std::cout<<"Copy result back"<<std::endl;
    std::array<float,64> squareEntropy;
    CHECK(hipMemcpy(squareEntropy.data(),squareEntropyIn,64*sizeof(float),hipMemcpyDeviceToHost));
    
    std::array<float,36> scanSquareEntropy;
    CHECK(hipMemcpy(scanSquareEntropy.data(),scanSquareEntropyIn,36*sizeof(float),hipMemcpyDeviceToHost));
        
    hipFree(deviceDistribution);
    hipFree(deviceInfoSetPtr);
    hipFree(squareEntropyIn);
    hipFree(scanSquareEntropyIn);
    hipFree(squareEntropyOut);
    hipFree(scanSquareEntropyOut);
    
    auto result = std::make_unique<std::pair<std::array<double,64>,std::array<double,36>>>();
    for(uint i=0; i<result->first.size(); i++)
        result->first[i] = squareEntropy[i];
    for(uint i=0; i<result->second.size(); i++)
        result->second[i] = scanSquareEntropy[i];
    
    return result;
}
}
