#include "hip/hip_runtime.h"
#include "chessinformationset.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

namespace crazyara {

void CHECK(hipError_t cuError)
{
    if(cuError!=hipSuccess)
    {   
        std::string cudaErrorString(hipGetErrorString(cuError));
        std::cout<<"CUDA Error: "<<cudaErrorString<<std::endl;
        throw std::logic_error("CUDA Fail");
    }
}

__device__ bool evaluateLiteral
(
    uint8_t* literal,
    uint8_t* board
)
{
    uint8_t sum = 0;
    for(uint8_t byteInd=0; byteInd<48; byteInd++)
    {
        sum += *(literal+byteInd) & *(board+byteInd);
        //printf("lit:%d & board:%d = %d\n",*(literal+byteInd),*(board+byteInd),*(literal+byteInd) & *(board+byteInd));
    }
    return (sum!=0)?true:false;
}

__device__ bool CISgetBit
(
    uint8_t byte,
    uint8_t position
)
{
    return byte & (1 << position);
}

__global__ void checkConditions
(
    uint8_t numberOfConditions,
    uint8_t* clausesPerCondition,
    uint8_t* conditionArray,
    uint8_t* boardInfoSet,
    uint64_t boardInfoSetSize,
    uint8_t* incompatibleBoards    
)
{
    __shared__ uint8_t clauseNbr[25];
    __shared__ uint8_t clauses[25][2][49];
    uint8_t* clauseStart = conditionArray;
    if(threadIdx.x==0)
    {
        for(uint condInd=0; condInd<numberOfConditions; condInd++)
        {
            uint8_t numberOfClauses = clausesPerCondition[condInd];
            clauseNbr[condInd] = numberOfClauses;
            for(uint clauseInd=0; clauseInd<numberOfClauses; clauseInd++)
            {
                for(uint byteInd=0; byteInd<49; byteInd++)
                {
                    clauses[condInd][clauseInd][byteInd] = *(clauseStart+byteInd);
                }
                clauseStart += 49;
            }
        }
    }
    __syncthreads();
    
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < boardInfoSetSize)
    {
        uint8_t board[58];
        for(uint8_t byteInd=0; byteInd<58; byteInd++)
        {
            board[byteInd] = boardInfoSet[index*58+byteInd];
        }
        
        bool compatible = true;
        for(uint8_t condInd=0; condInd<numberOfConditions; condInd++)
        {
            bool oneClauseTrue = false;
            uint8_t thisCondClauseNbr = clauseNbr[condInd];
            //printf("oneClauseTrue %d\n",oneClauseTrue);
            // IMPORTANT: No clause can be be empty
            for(uint8_t clauseInd=0; clauseInd<thisCondClauseNbr; clauseInd++)
            {
                bool clauseDemandBool = (bool)clauses[condInd][clauseInd][0];
                uint8_t* clauseBits = clauses[condInd][clauseInd]+1;
                bool literalBool = evaluateLiteral(clauseBits,board);
                oneClauseTrue |= (clauseDemandBool==literalBool);
                //printf("oneClauseTrue %d\n",oneClauseTrue);
            }
            compatible &= oneClauseTrue;
        }
        //printf("compatible %d\n",compatible);
        incompatibleBoards[index] = (compatible)?1:0;
    }
}

void ChessInformationSet::markIncompatibleBoardsGPU
(
    const std::vector<BoardClause>& conditions
)
{
    std::cout<<"Mark boards that do not fit: ";
    for(auto clause : conditions)
        std::cout<<clause.to_string()<<"&&";
    std::cout<<std::endl;
    
    std::unique_ptr<std::vector<std::uint8_t>> incompatibleBoard = checkBoardsValidGPU(conditions);
    // This is wrong
    std::for_each(incompatibleBoard->begin(),incompatibleBoard->end(),
                  [&](std::uint8_t boardIndex){incompatibleBoards.push(boardIndex);});
}

std::unique_ptr<std::vector<std::uint8_t>> ChessInformationSet::checkBoardsValidGPU
(
    const std::vector<BoardClause>& conditions
)
{
    /*
    std::cout<<"Mark boards that do not fit: ";
    for(auto clause : conditions)
        std::cout<<clause.to_string()<<"&&";
    std::cout<<std::endl;
    */
    
    std::vector<std::vector<std::pair<std::uint8_t,std::array<std::uint8_t,48>>>> hostBitwiseCondition;
    hostBitwiseCondition.resize(conditions.size());
    std::uint8_t numberOfConditions = conditions.size();
    std::vector<std::uint8_t> hostClausesPerCondition(numberOfConditions);
    if(numberOfConditions>25)
        throw std::logic_error("There must maximal 25 conditions");
    for(uint conditionInd=0; conditionInd<numberOfConditions; conditionInd++)
    {
        conditions[conditionInd].to_bits(hostBitwiseCondition[conditionInd]);
        hostClausesPerCondition[conditionInd] = hostBitwiseCondition[conditionInd].size();
        if(hostBitwiseCondition[conditionInd].size()>2)
            throw std::logic_error("There must only be 2 clauses per condition");
    }
    
    //std::cout<<"numberOfConditions:"<<int(numberOfConditions)<<std::endl;
    
    uint8_t* deviceClausesPerCondition;
    CHECK(hipMalloc((void**)&deviceClausesPerCondition,numberOfConditions*sizeof(uint8_t)));
    CHECK(hipMemcpy(deviceClausesPerCondition,hostClausesPerCondition.data(),
                         numberOfConditions*sizeof(uint8_t),hipMemcpyHostToDevice));
    
    //std::cout<<"Start remapping"<<std::endl;
    std::vector<std::uint8_t> remapClausesPerCondition(numberOfConditions);
    CHECK(hipMemcpy(remapClausesPerCondition.data(),deviceClausesPerCondition,
                         numberOfConditions*sizeof(uint8_t),hipMemcpyDeviceToHost));
    //std::for_each(remapClausesPerCondition.begin(),remapClausesPerCondition.end(),[](auto ind){std::cout<<int(ind)<<" ";});
    //std::cout<<"Remapped"<<std::endl;
    
    std::vector<std::pair<std::uint8_t,std::array<std::uint8_t,48>>> hostBitwiseConditionFlat;
    for(auto oneCondition : hostBitwiseCondition)
    {
        /*
        for(auto oneClause : oneCondition)
        {
            auto firstPart = oneClause.first;
            for(auto byte : firstPart)
                std::cout<<uint(byte)<<" ";
            std::cout<<std::endl;
            auto secondPart = oneClause.second;
            for(auto byte : secondPart)
                std::cout<<uint(byte)<<" ";
            std::cout<<std::endl;
        }
        */
        hostBitwiseConditionFlat.insert(hostBitwiseConditionFlat.end(),oneCondition.begin(),oneCondition.end());
    }
    
    uint8_t* deviceBitwiseCondition;
    uint byteSizeMem = hostBitwiseConditionFlat.size()*sizeof(uint8_t)*49;
    CHECK(hipMalloc((void**)&(deviceBitwiseCondition),byteSizeMem));
    CHECK(hipMemcpy(deviceBitwiseCondition,hostBitwiseConditionFlat.data(),byteSizeMem,hipMemcpyHostToDevice));    

    //std::cout<<"Start remapping"<<std::endl;
    std::vector<std::pair<std::uint8_t,std::array<std::uint8_t,48>>> remapHostBitwiseCondition(hostBitwiseConditionFlat.size());
    CHECK(hipMemcpy(remapHostBitwiseCondition.data(),deviceBitwiseCondition,byteSizeMem,hipMemcpyDeviceToHost));    
    
    /*
    for(auto oneCondition : remapHostBitwiseCondition)
    {
        std::cout<<"Cond:"<<std::endl;
        auto firstPart = oneCondition.first;
        std::cout<<" Must be: ";
        for(auto byte : firstPart)
            std::cout<<uint(byte)<<" ";
        std::cout<<std::endl;
        auto secondPart = oneCondition.second;
        std::cout<<" Must not be: ";
        for(auto byte : secondPart)
            std::cout<<uint(byte)<<" ";
        std::cout<<std::endl;
    }
    std::cout<<"Remapped"<<std::endl;
    */
    

    
    std::uint64_t cis_size = size();
    std::uint64_t cis_byte_size = cis_size*(chessInfoSize/8);
    std::uint8_t* hostInfoSetPtr = getInfoSetPtr();
    uint8_t* deviceInfoSetPtr;
    CHECK(hipMalloc((void**)&deviceInfoSetPtr,cis_byte_size*sizeof(uint8_t)));
    CHECK(hipMemcpy(deviceInfoSetPtr,hostInfoSetPtr,cis_byte_size*sizeof(uint8_t),hipMemcpyHostToDevice));
    
    //std::cout<<"cis_size:"<<int(cis_size)<<std::endl;
    //std::cout<<"cis_byte_size:"<<int(cis_byte_size)<<std::endl;
    
    auto result = std::make_unique<std::vector<std::uint8_t>>(cis_size);
    std::vector<std::uint8_t>& hostIncompatibleBoards = *result;
    uint8_t* deviceIncompatibleBoards;
    CHECK(hipMalloc((void**)&deviceIncompatibleBoards,cis_size*sizeof(uint8_t)));
    
    //std::cout<<"Before kernel invocation"<<std::endl;
    
    int suggested_blockSize; 
    int suggested_minGridSize;
    hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, checkConditions, 0, 0);
    int device;
    hipGetDevice(&device); 
    struct hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);    
    
    //std::cout<<"suggested_blockSize:"<<int(suggested_blockSize)<<std::endl;
    //std::cout<<"suggested_minGridSize:"<<int(suggested_minGridSize)<<std::endl;
    //std::cout<<"device:"<<int(device)<<std::endl;

    dim3 blocks(suggested_blockSize);
    //std::cout<<"blocks.x:"<<blocks.x<<std::endl;
    dim3 grids(ceil((float)cis_size/suggested_blockSize));
    //std::cout<<"grids.x:"<<grids.x<<std::endl;
    
    checkConditions<<<grids,blocks>>>
    (
        numberOfConditions,
        deviceClausesPerCondition,
        deviceBitwiseCondition,
        deviceInfoSetPtr,
        cis_size,
        deviceIncompatibleBoards
    );
    
    //std::cout<<"After kernel invocation"<<std::endl;
    
    CHECK(hipMemcpy(hostIncompatibleBoards.data(),deviceIncompatibleBoards,
                     cis_size*sizeof(uint8_t),hipMemcpyDeviceToHost));
    
    hipFree(deviceClausesPerCondition);
    hipFree(deviceBitwiseCondition);
    hipFree(deviceInfoSetPtr);
    hipFree(deviceIncompatibleBoards);

    for(uint64_t boardIndex=0; boardIndex<hostIncompatibleBoards.size(); boardIndex++)
    {
        if(hostIncompatibleBoards[boardIndex]==1)
            incompatibleBoards.push(boardIndex);
    }
    
    //std::cout<<"End of function"<<std::endl;

    return result;
}

__global__ void initialReduce // blockDim.x == 32
(
    uint8_t* boardInfoSet,
    uint64_t boardInfoSetSize,
    uint32_t* boardSum //gridsize * (64*6)
)
{
    __shared__ uint32_t distro[32][6][64];
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        for(uint8_t squareInd=0; squareInd<64; squareInd++)
        {
            distro[threadIdx.x][pieceInd][squareInd] = 0;
        }
    }
    __syncthreads();
    uint64_t boardSize = 58;
    uint64_t blockSpan = ceilf((float)boardInfoSetSize / gridDim.x);
    uint64_t blockOffset = blockIdx.x*blockSpan;
    
    // Reduce to 32 boards shared memory
    uint8_t* blockStartPtr = boardInfoSet+blockOffset*boardSize;
    uint64_t validBlockSpan = min(blockSpan,boardInfoSetSize-blockOffset);
    for(uint64_t locIndex=threadIdx.x; locIndex<validBlockSpan; locIndex+=blockDim.x)
    {
        uint8_t* boardStart = blockStartPtr + locIndex*boardSize;
        uint8_t* probabilityPtr = boardStart+57;
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            uint8_t* pieceBoardStart = boardStart + pieceInd*8;
            uint8_t adding = 1; //*probabilityPtr & 127; // unset first bit
            for(uint8_t row=0; row<8; row++)
            {
                uint8_t* boardRow = pieceBoardStart + row;
                for(uint8_t col=0; col<8; col++)
                {
                    bool squareOccupied = CISgetBit(*boardRow,7-col);
                    distro[threadIdx.x][pieceInd][row*8+col] += (squareOccupied)?adding:0; 
                }
            }
        }
    }
    __syncthreads();
    
    for(uint8_t size=1; size<32; size++)
    {
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            distro[0][pieceInd][threadIdx.x] += distro[size][pieceInd][threadIdx.x];
            distro[0][pieceInd][threadIdx.x+32] += distro[size][pieceInd][threadIdx.x+32];
        }
    }
    __syncthreads();
    
    uint32_t* boardSumOffset = boardSum + blockIdx.x*(64*6);
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        *(boardSumOffset + (pieceInd*64) + threadIdx.x) = distro[0][pieceInd][threadIdx.x];
        *(boardSumOffset + (pieceInd*64) + threadIdx.x+32) = distro[0][pieceInd][threadIdx.x+32];
    }
}

__global__ void reduce // blockDim.x == 32
(
    uint32_t* boardsIn,
    uint32_t boardInSize,
    uint32_t* boardsOut //gridsize * (64*6)
)
{
    __shared__ uint32_t distro[32][6][64];
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        for(uint8_t squareInd=0; squareInd<64; squareInd++)
        {
            distro[threadIdx.x][pieceInd][squareInd] = 0;
        }
    }
    uint64_t blockSpan = ceilf((float)boardInSize / gridDim.x);
    uint64_t blockOffset = blockIdx.x*blockSpan;
    uint64_t blockOffsetData = blockOffset*6*64;

    for(uint64_t locIndex = 0; locIndex+threadIdx.x<blockSpan; locIndex+=blockDim.x)
    {
        uint32_t* boardStart = boardsIn + blockOffsetData + (locIndex+threadIdx.x)*6*64;
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            for(uint8_t squareInd=0; squareInd<64; squareInd++)
            {
                distro[threadIdx.x][pieceInd][squareInd] += *(boardStart+pieceInd*64+squareInd);
            }
        }
    }
    
    __syncthreads();
    for(uint8_t size=1; size<32; size++)
    {
        for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
        {
            distro[0][pieceInd][threadIdx.x] += distro[size][pieceInd][threadIdx.x];
            distro[0][pieceInd][threadIdx.x+32] += distro[size][pieceInd][threadIdx.x+32];
        }
    }
    
    __syncthreads();
    uint32_t* boardSumOffset = boardsOut + blockIdx.x*(64*6);
    for(uint8_t pieceInd=0; pieceInd<6; pieceInd++)
    {
        *(boardSumOffset + (pieceInd*64) + threadIdx.x) = distro[0][pieceInd][threadIdx.x];
        *(boardSumOffset + (pieceInd*64) + threadIdx.x+32) = distro[0][pieceInd][threadIdx.x+32];
    }
}

std::unique_ptr<ChessInformationSet::Distribution> ChessInformationSet::computeDistributionGPU()
{
    //std::cout<<"Compute Distribution"<<std::endl;
    std::uint64_t cis_size = size();
    std::uint64_t maxSize = 1;
    maxSize = maxSize<<32;
    if(cis_size >= maxSize)
    {
        std::cout<<"cis_size:"<<cis_size<<" > "<<maxSize<<std::endl;
        throw std::invalid_argument("CIS size too big");
    }
    std::uint64_t cis_byte_size = cis_size*(chessInfoSize/8);
    std::uint8_t* hostInfoSetPtr = getInfoSetPtr();
    uint8_t* deviceInfoSetPtr;
    CHECK(hipMalloc((void**)&deviceInfoSetPtr,cis_byte_size*sizeof(uint8_t)));
    CHECK(hipMemcpy(deviceInfoSetPtr,hostInfoSetPtr,cis_byte_size*sizeof(uint8_t),hipMemcpyHostToDevice));
    
    /*
    for(int i=0; i<cis_size; i++)
    {
        for(int k=0; k<58; k++)
            std::cout<<int(hostInfoSetPtr[i*58+k])<<" ";
        std::cout<<std::endl;
    }
    std::cout<<std::endl;
    */
        
    //std::cout<<"cis_size:"<<int(cis_size)<<std::endl;
    //std::cout<<"cis_byte_size:"<<int(cis_byte_size)<<std::endl;
    
    dim3 blocks(32);
    
    std::uint64_t maxNbrBlocks = 60000;
    std::uint64_t boardsPerBlock = cis_size / maxNbrBlocks;
    std::uint64_t boardsPerThread = boardsPerBlock / blocks.x;
    std::uint64_t minBoardsPerThread = 8;
    if(boardsPerThread < minBoardsPerThread)
        boardsPerThread = minBoardsPerThread;    
    dim3 grids(ceil((float)cis_size/(blocks.x*boardsPerThread)));

    //std::cout<<"blocks.x:"<<blocks.x<<std::endl;
    //std::cout<<"grids.x:"<<grids.x<<std::endl;
    
    uint32_t* deviceBoardSumIn;
    CHECK(hipMalloc((void**)&deviceBoardSumIn,grids.x*6*64*sizeof(uint32_t)));
    
    uint32_t* deviceBoardSumOut;
    CHECK(hipMalloc((void**)&deviceBoardSumOut,grids.x*6*64*sizeof(uint32_t)));
    
    initialReduce<<<grids,blocks>>>
    (
        deviceInfoSetPtr,
        cis_size,
        deviceBoardSumIn
    );
    hipDeviceSynchronize();
    //std::cout<<"Initial reduction"<<std::endl;
    
    std::uint64_t inGridSize = grids.x;
    std::uint64_t outGridSize;
    while(inGridSize>1)
    {
        outGridSize = ceil((float)inGridSize / 128);
        grids = dim3(outGridSize);
        //std::cout<<"inGridSize:"<<inGridSize<<std::endl;
        //std::cout<<"outGridSize:"<<outGridSize<<std::endl;
        reduce<<<grids,blocks>>>
        (
            deviceBoardSumIn,
            inGridSize,
            deviceBoardSumOut
        );
        hipDeviceSynchronize();
        inGridSize = outGridSize;
        uint32_t* temp = deviceBoardSumIn;
        deviceBoardSumIn = deviceBoardSumOut;
        deviceBoardSumOut = temp;
    }
    //std::cout<<"Copy result back"<<std::endl;
    std::array<std::uint32_t,384> piecesSum;
    CHECK(hipMemcpy(piecesSum.data(),deviceBoardSumIn,384*sizeof(uint32_t),hipMemcpyDeviceToHost));
    
    /*
    for(auto count : piecesSum)
        std::cout<<count<<" ";
    std::cout<<std::endl;
    */
    
    //std::cout<<"Free"<<std::endl;
    hipFree(deviceInfoSetPtr);
    hipFree(deviceBoardSumIn);
    hipFree(deviceBoardSumOut);
    
    //std::cout<<"Compute Fraction"<<std::endl;
    std::array<double,384> piecesSumDouble;
    for(uint i=0; i<piecesSum.size(); i++)
        piecesSumDouble[i] = static_cast<double>(piecesSum[i]) / cis_size;
    
    //std::cout<<"Compute Distribution"<<std::endl;
    auto piecesDistro = std::make_unique<Distribution>();
    std::memcpy(piecesDistro->pawns.data(),  piecesSumDouble.data(),    64*sizeof(double));
    std::memcpy(piecesDistro->knights.data(),piecesSumDouble.data()+64, 64*sizeof(double));
    std::memcpy(piecesDistro->bishops.data(),piecesSumDouble.data()+128,64*sizeof(double));
    std::memcpy(piecesDistro->rooks.data(),  piecesSumDouble.data()+192,64*sizeof(double));
    std::memcpy(piecesDistro->queens.data(), piecesSumDouble.data()+256,64*sizeof(double));
    std::memcpy(piecesDistro->kings.data(),  piecesSumDouble.data()+320,64*sizeof(double));
    //std::cout<<"Return"<<std::endl;
    return piecesDistro;
}
}
